#include "hip/hip_runtime.h"
#include "src/settings.h"
#include <stdio.h>
#include <time.h>

extern "C"{
#include "src/helpers.h"
#include "src/cpuModel.h"
}

int main(int argc, char* argv[]){
    if(argc != 4){
        fprintf(stderr, "Usage: ./nbody cat.csv framesAmount writeRate\n");
        exit(1);
    };
    
    frame* test = readFrame(argv[1]);
    int pathLen = sizeof("out/out000000.csv");
    char path[pathLen];
   
    double ** m = prepareGravitationalParameters(test->masses);
    
    int FRAMES_AMOUNT = atoi(argv[2]);
    if(FRAMES_AMOUNT < 1){
        fprintf(stderr, "ERROR: Too small frames amount\n");
        exit(2);
    };
    int WRITE_STEP = atoi(argv[3]);
    if(WRITE_STEP < 1){
        fprintf(stderr, "ERROR: Too small writing step\n");
        exit(1);
    };
    
    unsigned int start = clock();
    
    for(int i = 0; i < FRAMES_AMOUNT; i++){
        for(int j = 0; j < WRITE_STEP; j++){
            updateFrame(test, (const double **)m);
        };
        if(sprintf(path, "out/out%06d.csv", i) != pathLen - 1){
            fprintf(stderr, "ERROR: Can't generate filename\n");
            fprintf(stderr, "PathLen: %d\n", pathLen);
            exit(1);
        };
        path[pathLen - 1] = '\0';
        writeFrameShort(path, test);
        fprintf(stdout, "Frame#%05d created\n", i);
    };
    unsigned int stop = clock();
    double avg = (double)(stop - start)/ CLOCKS_PER_SEC;
    avg/=FRAMES_AMOUNT;
    fprintf(stdout, "Average time per frame: %f s\n", avg);
    
    writeFrameFull("result.csv", test);
   
    freeSquareMatrix(m);
    freeFrame(test);
    free(test);
    return 0;
};
